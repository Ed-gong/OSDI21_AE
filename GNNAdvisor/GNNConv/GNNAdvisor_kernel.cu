#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <vector>
#define MAX_DIM 30
#define MAX_NB 32       // must <= partsize 
#define threadPerWarp 32 //must < 32
#define wrapPerBlock 8  // must also set with respect to the 
                        // [thread-per-block = wrapPerBlock*threadPerWarp]

__device__ inline float atomicAdd_F(float* address, float value)
{
  float old = value;  
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
}

template <typename scalar_t>
__global__ void spmm_forward_cuda_kernel(
    const int num_nodes, 
    const int dim,
    const int num_parts,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> row_pointers, 
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part2Node
);

template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel(
    const int num_nodes, 
    const int dim,
    const int num_parts,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part2Node
);


////////////////////////////////////////////
//
// Foward Pass
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_forward_cuda(
    torch::Tensor input,
    torch::Tensor weight,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    torch::Tensor degrees,
    torch::Tensor part_pointers,
    torch::Tensor part2Node,
    int threadPerBlock
) 
{
    auto tmp = torch::mm(input, weight);
    auto output = torch::zeros_like(tmp);

    const int dim = tmp.size(1);
    const int num_nodes = tmp.size(0);
    const int num_parts = part2Node.size(0);

    const int block_size = wrapPerBlock * threadPerWarp;
    const int blocks = (num_parts * 32 + block_size  - 1) / block_size; 

    AT_DISPATCH_FLOATING_TYPES(input.type(), "spmm_cuda_forward", ([&] {
                                spmm_forward_cuda_kernel<scalar_t><<<blocks, threadPerBlock>>>(
                                    num_nodes, 
                                    dim,
                                    num_parts,
                                    tmp.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                    output.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                    row_pointers.packed_accessor32<int,1,torch::RestrictPtrTraits>(), 
                                    column_index.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
                                    degrees.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
                                    part_pointers.packed_accessor32<int,1,torch::RestrictPtrTraits>(), 
                                    part2Node.packed_accessor32<int,1,torch::RestrictPtrTraits>()
                                );
                            }));
                                 
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    return {output};
}

template <typename scalar_t>
__global__ void spmm_forward_cuda_kernel(
    const int num_nodes, 
    const int dim,
    const int num_parts, 
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> row_pointers, 
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part2Node
) {

    int tid =  blockIdx.x * blockDim.x + threadIdx.x;
    int warpId =  tid / 32;                             // global warp-id
    int block_warpID = threadIdx.x/32;                  // block warp-id
    int intraWarp_tid = tid % 32;                       // warp thread-id

    if (warpId < num_parts && intraWarp_tid < threadPerWarp){

        __shared__  int partial_index[MAX_NB * wrapPerBlock];
        __shared__ float partial_results[MAX_DIM * wrapPerBlock];

        int srcId = part2Node[warpId];
        int partBeg = part_pointers[warpId];
        int partEnd = part_pointers[warpId + 1];
        float src_norm = degrees[srcId];

        int pindex_base = block_warpID * MAX_NB;
        for (int nid = partBeg + intraWarp_tid; nid < partEnd; nid += threadPerWarp){
            partial_index[pindex_base + nid - partBeg] = column_index[nid];
        }
         __syncthreads();

        int presult_base = block_warpID * MAX_DIM;
        for (int nid = 0; nid < partEnd - partBeg; nid++)
        {
            int nIndex = partial_index[pindex_base + nid];
            float degree_norm_inv = __fmaf_rn(src_norm, degrees[nIndex], 0);

            if (nid == 0)
                #pragma unroll
                for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
                    partial_results[presult_base + d] = 0;
                }
            
            #pragma unroll
            for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
                partial_results[presult_base + d] += __fmaf_rn(degree_norm_inv, input[nIndex][d], 0);
            }
        }

        #pragma unroll
        for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
            atomicAdd_F((float*)&output[srcId][d], partial_results[presult_base + d]);
        }
    }

    // if (tid < num_nodes){
    //     for(int nid = row_pointers[tid]; nid < row_pointers[tid + 1]; nid++){
    //         int nIndex = column_index[nid];
    //         float degree_norm_inv = 1.0/sqrt(degrees[tid]) * (1.0/sqrt(degrees[nIndex]));
    //         for (int d = 0; d < dim; d++){
    //             output[tid][d] += degree_norm_inv * input[nIndex][d];
    //         }
    //     }
    // }
}

////////////////////////////////////////////
// 
// backward pass
//
////////////////////////////////////////////
std::vector<torch::Tensor> spmm_backward_cuda(
    torch::Tensor d_output,
    torch::Tensor X,
    torch::Tensor W,
    torch::Tensor row_pointers,
    torch::Tensor column_index,
    torch::Tensor degrees,
    torch::Tensor part_pointers,
    torch::Tensor part2Node,
    int threadPerBlock
) {

    auto d_input_prime = torch::zeros_like(d_output);

    const int dim = d_input_prime.size(1);
    const int num_nodes = d_input_prime.size(0);
    const int num_parts = part2Node.size(0);

    const int block_size = wrapPerBlock * threadPerWarp;
    const int blocks = (num_parts * 32 + block_size - 1) / block_size; 

    AT_DISPATCH_FLOATING_TYPES(d_output.type(), "spmm_cuda_backward", ([&] {
                                spmm_backward_cuda_kernel<scalar_t><<<blocks, block_size>>>(
                                    num_nodes, 
                                    dim,
                                    num_parts,
                                    d_output.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                    d_input_prime.packed_accessor32<scalar_t,2,torch::RestrictPtrTraits>(),
                                    row_pointers.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
                                    column_index.packed_accessor32<int,1,torch::RestrictPtrTraits>(),
                                    degrees.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
                                    part_pointers.packed_accessor32<int,1,torch::RestrictPtrTraits>(), 
                                    part2Node.packed_accessor32<int,1,torch::RestrictPtrTraits>()
                                );
                            }));
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    auto d_input = torch::mm(d_input_prime, W.transpose(0,1));
    auto d_weight = torch::mm(X.transpose(0,1), d_input_prime);
    // d_input = torch::mm(d_input_prime, W.transpose(0,1))
    // d_weights = torch::mm(X.transpose(0,1), d_input_prime)
    return {d_input, d_weight};
}

template <typename scalar_t>
__global__ void spmm_backward_cuda_kernel(
    const int num_nodes, 
    const int dim,
    const int num_parts, 
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> d_input,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> row_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> column_index,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> degrees,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part_pointers,
    torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> part2Node
) {

    int tid =  blockIdx.x * blockDim.x + threadIdx.x;
    int warpId =  tid / 32;
    int intraWarp_tid = tid % 32;
    int block_warpID = threadIdx.x/32;
    
    if (warpId < num_parts && intraWarp_tid < threadPerWarp){

        __shared__  int partial_index[MAX_NB * wrapPerBlock];
        __shared__ float partial_results[MAX_DIM * wrapPerBlock];
        // __shared__ float dst_norm[MAX_NB * wrapPerBlock];

        int srcId = part2Node[warpId];
        int partBeg = part_pointers[warpId];
        int partEnd = part_pointers[warpId + 1];
        float src_norm = degrees[srcId];

        int pindex_base = block_warpID * MAX_NB;
        for (int nid = partBeg + intraWarp_tid; nid < partEnd; nid += threadPerWarp){
            partial_index[pindex_base + nid - partBeg] = column_index[nid];
            // dst_norm[pindex_base + nid - partBeg] = src_norm * degrees[column_index[nid]];
        }
         __syncthreads();

        int presult_base = block_warpID * MAX_DIM;
        for (int nid = 0; nid < partEnd - partBeg; nid++)
        {
            int nIndex = partial_index[pindex_base + nid];
            float degree_norm =  __fmaf_rn(src_norm, degrees[nIndex], 0);

            if (nid == 0)
                #pragma unroll
                for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
                    partial_results[presult_base + d] = 0;
                    // atomicAdd_F((float*)&d_input[srcId][d], degree_norm * d_output[nIndex][d]);
                }
            
                #pragma unroll
            for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
                partial_results[presult_base + d] += __fmaf_rn(degree_norm, d_output[nIndex][d], 0);
            }
        }
        for (int d = intraWarp_tid; d < dim; d += threadPerWarp){
            atomicAdd_F((float*)&d_input[srcId][d], partial_results[presult_base + d]);
        }
    }
    // int tid =  blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < num_nodes){
    //     for(int nid = row_pointers[tid]; nid < row_pointers[tid + 1]; nid++){
    //         int nIndex = column_index[nid];
    //         float degree_norm = sqrt(degrees[tid]) * sqrt(degrees[nIndex]);
    //         for (int d = 0; d < dim; d++){
    //             d_input[tid][d] += degree_norm * d_output[nIndex][d];
    //         }
    //     }
    // }
}
